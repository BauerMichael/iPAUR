#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
// #define CAMERA

inline __device__ float apply_gamma(float x, float gamma) {
    return pow(x, gamma);
}

__global__ void gamma_correction(float* out, float* in, float gamma, int width, int height, int pitch, int channel) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < width && y < height) {
        for (int c = 0; c < channel; c++) {
            float* row_a = (float*)((char*)in + (y + c * height) * pitch);
            float* out_a = (float*)((char*)out + (y + c * height) * pitch);
            out_a[x] = apply_gamma(row_a[x], gamma);
        }
    }
}

void gamma_correction_cpu(float* out, float* in, float gamma, int size) {
    for (int i = 0; i < size; i++) {
        out[i] = pow(in[i], gamma);
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed
    float gamma = 1.0;
    getParam("gamma", gamma, argc, argv);
    cout << "gamma: " << gamma << endl;

    // ### Define your own parameters here as needed
    bool gpu = false;
    getParam("gpu", gpu, argc, argv);
    cout << "gpu: " << gpu << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
    cv::VideoCapture camera(0);
    if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
    camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
    camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int size = w * h * nc;
    // int nbyte = size * sizeof(float);
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *h_imgIn  = new float[size];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *h_imgOut = new float[size];

    // allocate raw input image for GPU
    float* d_imgIn;
    float* d_imgOut;

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (h_imgIn, mIn);

    unsigned long int pitch;
    unsigned long int memsize = w * sizeof(float);
    hipMallocPitch(&d_imgIn, &pitch, memsize, h*nc);
    hipMallocPitch(&d_imgOut, &pitch, memsize, h*nc);

    // copy host memory
    hipMemcpy2D(d_imgIn, pitch, h_imgIn, memsize, memsize, h*nc, hipMemcpyHostToDevice);
    CUDA_CHECK;
    // hipMemcpy(d_imgIn, h_imgIn, nbyte, hipMemcpyHostToDevice);
    // CUDA_CHECK;

    // launch kernel
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);

    Timer timer; timer.start();

    for (int i = 0; i < repeats; i++) {
        if (gpu) {
            gamma_correction <<<grid, block>>> (d_imgOut, d_imgIn, gamma, w, h, pitch, nc);
            hipDeviceSynchronize();
        } else {
            gamma_correction_cpu(h_imgOut, h_imgIn, gamma, size);
        }
    }

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

    hipMemcpy2D(h_imgOut, memsize, d_imgOut, pitch, memsize, h*nc, hipMemcpyDeviceToHost);
    CUDA_CHECK;

    // hipMemcpy(h_imgOut, d_imgIn, nbyte, hipMemcpyDeviceToHost);
    // CUDA_CHECK;

    // free GPU memory
    hipFree(d_imgIn);
    CUDA_CHECK;
    hipFree(d_imgOut);
    CUDA_CHECK;


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, h_imgOut);
    showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] h_imgIn;
    delete[] h_imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
