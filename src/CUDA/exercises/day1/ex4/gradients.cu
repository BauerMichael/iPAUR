#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###


#include "aux.h"
#include <iostream>
using namespace std;

// uncomment to use the camera
// #define CAMERA

inline __device__ float add(float x1, float x2) {
    return x1 + x2;
}

__global__ void addArray(float* out, float* in1, float* in2, int size) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < size) {
        out[index] = add(in1[index], in2[index]);
    }
}

__global__ void l2(float* out, float* in, int width, int height, int channel) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int index = x + width * y;
    float sum = 0.0;
    if (x < width && y < height) {
        for (int i = 0; i < channel; i++) {
            sum += pow(in[index + i * height * width], 2);
        }
        out[index] = sqrtf(sum);
    }
}

__global__ void del_x_plus(float* out, float* in, int width, int height) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    int index = x + width * y + width * height * c;
    if (x < width && y < height) {
        out[index] = x < width + 1 ? in[x+1 + width * y + width * height * c] - in[index] : 0.f;
    }
}

__global__ void del_y_plus(float* out, float* in, int width, int height) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    int index = x + width * y + width * height * c;
    if (x < width && y < height) {
        out[index] = y < height + 1 ? in[x + width * (y+1) + width * height * c] - in[index] : 0.f;
    }
}

__global__ void del_x_minus(float* out, float* in, int width, int height) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    int index = x + width * y + width * height * c;
    if (x < width && y < height) {
        out[index] = x > 0 ? in[index] - in[x-1 + y * width + width * height * c] : in[index];
    }
}

__global__ void del_y_minus(float* out, float* in, int width, int height) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int c = threadIdx.z + blockDim.z * blockIdx.z;
    int index = x + width * y + width * height * c;
    if (x < width && y < height) {
        out[index] = y > 0 ? in[index] - in[x + (y-1) * width + width * height * c] : in[index];
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int size = w * h * nc;
    int nbyte = size * sizeof(float);
    cout << "image: " << w << " x " << h << endl;

    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    cv::Mat mOut2(h, w, CV_8UC1);  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed

    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *h_imgIn  = new float[(size_t)size];
    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
    float *h_imgOut = new float[(size_t)w*h*mOut.channels()];
    float *h_absum = new float[(size_t)w*h];

    // allocate raw input image for GPU
    float* d_imgIn;
    float* d_imgOut1;
    float* d_imgOut2;
    float* d_imgOut3;
    float* d_sum;
    float* d_absum;

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (h_imgIn, mIn);

    // alloc GPU memory
    hipMalloc(&d_imgIn, nbyte);
    CUDA_CHECK;
    hipMalloc(&d_imgOut1, nbyte);
    CUDA_CHECK;
    hipMalloc(&d_imgOut2, nbyte);
    CUDA_CHECK;
    hipMalloc(&d_imgOut3, nbyte);
    CUDA_CHECK;
    hipMalloc(&d_sum, nbyte);
    CUDA_CHECK;
    hipMalloc(&d_absum, w*h*sizeof(float));
    CUDA_CHECK;

    // copy host memory
    hipMemcpy(d_imgIn, h_imgIn, nbyte, hipMemcpyHostToDevice);
    CUDA_CHECK;

    // launch kernel
    dim3 block = dim3(32, 8, nc);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (nc + block.z - 1) / block.z);

    dim3 block_sum_up = dim3(256, 1, 1);
    dim3 grid_sum_up = dim3((size + block_sum_up.x - 1) / block_sum_up.x, 1, 1);

    dim3 block_l2 = dim3(32, 8, 1);
    dim3 grid_l2 = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);

    Timer timer; timer.start();

    for (int i = 0; i < repeats; i++) {
        del_x_plus <<<grid, block>>> (d_imgOut1, d_imgIn, w, h);
        del_y_plus <<<grid, block>>> (d_imgOut2, d_imgIn, w, h);
        del_x_minus <<<grid, block>>> (d_imgOut3, d_imgOut1, w, h);
        del_y_minus <<<grid, block>>> (d_imgOut1, d_imgOut2, w, h);
        addArray <<<grid_sum_up, block_sum_up>>> (d_sum, d_imgOut1, d_imgOut3, size);
        l2 <<<grid_l2, block_l2 >>> (d_absum, d_sum, w, h, nc);
    }

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time: " << t*1000 << " ms" << endl;

    hipMemcpy(h_absum, d_absum, w*h*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK;
    hipMemcpy(h_imgOut, d_sum, nbyte, hipMemcpyDeviceToHost);
    CUDA_CHECK;

    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
        {
            mOut2.at<uchar>(i, j) = h_absum[j + i * w]*255;
        }
    }

    // free GPU memory
    hipFree(d_imgIn);
    CUDA_CHECK;
    hipFree(d_sum);
    CUDA_CHECK;
    hipFree(d_absum);
    CUDA_CHECK;
    hipFree(d_imgOut1);
    CUDA_CHECK;
    hipFree(d_imgOut2);
    CUDA_CHECK;
    hipFree(d_imgOut3);
    CUDA_CHECK;


    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    // convert_layered_to_mat(mOut, h_imgOut);
    showImage("Output", mOut2, 100+w+40, 100);
    // showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif

    // save input and result
    cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] h_imgIn;
    delete[] h_imgOut;
    delete[] h_absum;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}