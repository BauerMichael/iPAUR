#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include "aux.h"
#include <iostream>
#include <stdio.h>
using namespace std;

void parameterToFile(string filename,int repeats,bool gray,int level,float tau,float sigma,float lambda,float nu,int w,int h,int nc,size_t available,size_t total,float t,int iter) {
    FILE *file;
    file = fopen(filename.c_str(), "w");
    if(file == NULL)
        printf("ERROR: Could not open file!");
    else {
        fprintf(file, "image: %d x %d x %d\n", w, h, nc);
        fprintf(file,"repeats: %d\n", repeats);
        fprintf(file,"gray: %d\n", gray);
        fprintf(file,"level: %d\n", level);
        fprintf(file,"tau: %f\n", tau);
        fprintf(file,"sigma: %f\n", sigma);
        fprintf(file,"lambda: %f\n", lambda);
        fprintf(file,"nu: %f\n", nu);
        fprintf(file, "GPU Memory: %zd - %zd = %f GB\n", total, available, (total-available)/pow(10,9));
        fprintf(file, "time: %f s\n", t);
        fprintf(file, "iterations: %d\n", iter);
    }
    fclose (file);
}

void parameterToConsole(string filename,int repeats,bool gray,int level,float tau,float sigma,float lambda,float nu,int w,int h,int nc,size_t available,size_t total,float t,int iter) {
    printf( "image: %d x %d x %d\n", w, h, nc);
    printf("repeats: %d\n", repeats);
    printf("gray: %d\n", gray);
    printf("level: %d\n", level);
    printf("tau: %f\n", tau);
    printf("sigma: %f\n", sigma);
    printf("lambda: %f\n", lambda);
    printf("nu: %f\n", nu);
    printf( "GPU Memory: %zd - %zd = %f GB\n", total, available, (total-available)/pow(10,9));
    printf( "time: %f s\n", t);
    printf( "iterations: %d\n", iter);
}

__device__ float bound(float x1, float x2, float lambda, float k, float l, float f)
{
    return 0.25f * (x1*x1 + x2*x2) - lambda * pow(k / l - f, 2);
}

__device__ float interpolate(float k, float uk0, float uk1, float l)
{
    return (k + (0.5 - uk0) / (uk1 - uk0)) / l;
}

__device__ void on_parabola(float* u1, float* u2, float* u3, float x1, float x2, float x3, float f, float L, float lambda, float k, int j, float l)
{
    float y = x3 + lambda * pow(k / l - f, 2);
    float norm = sqrtf(x1*x1+x2*x2);
    float v = 0.f;
    float a = 2.f * 0.25f * norm;
    float b = 2.f / 3.f * (1.f - 2.f * 0.25f * y);
    float d = b < 0 ? (a - pow(sqrt(-b), 3)) * (a + pow(sqrt(-b), 3)) : a*a + b*b*b;
    float c = pow((a + sqrt(d)), 1.f/3.f);
    if (d >= 0) {
        v = c == 0 ? 0.f : c - b / c;
    } else {
        v = 2.f * sqrt(-b) * cos((1.f / 3.f) * acos(a / (pow(sqrt(-b), 3))));
    }
    u1[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x1 / norm;
    u2[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x2 / norm;
    u3[j] = bound(u1[j], u2[j], lambda, k, l, f);
}

__global__ void project_on_parabola(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float* img, float L, float lambda, int k, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (k-1) * w * h * l;
        u1[i] = u1[j];
        u2[i] = u2[j];
        u3[i] = u3[j];
    }

    if (x < w && y < h && z < l)
    {
        int index = x + w * y;
        int i = x + w * y + w * h * z + (k-1) * w * h * l;
        int j = x + w * y + w * h * z + k * w * h * l;

        float f = img[index];
        float x1 = u1[i] - v1[j];
        float x2 = u2[i] - v2[j];
        float x3 = u3[i] - v3[j];
        float bound_val = bound(x1, x2, lambda, (z+1.f), l, f);

        if (x3 < bound_val) {
            on_parabola(u1, u2, u3, x1, x2, x3, f, L, lambda, (z+1.f), j, l);
        } else {
            u1[j] = x1;
            u2[j] = x2;
            u3[j] = x3;
        }
    }

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z + k * w * h * l;
        int j = x + w * y + w * h * z + (k-1) * w * h * l;
        v1[i] = u1[i] - (u1[j] - v1[i]);
        v2[i] = u2[i] - (u2[j] - v2[i]);
        v3[i] = u3[i] - (u3[j] - v3[i]);
    }
}

__global__ void soft_shrinkage(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float nu, int k1, int k2, int P, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h)
    {
        int count_p = 2;
        float K;         
            
        for (int k1 = 0; k1 < l; k1++)
        {
            for (int k2 = k1; k2 < l; k2++)
            {
                K = (float)(k2 - k1 + 1);
                int i, j;
                float s1 = 0.f;
                float s2 = 0.f;
                float s01 = 0.f;
                float s02 = 0.f;
                float x1 = 0.f;
                float x2 = 0.f;

                for (int k = k1; k <= k2; k++)
                {
                    i = x + w * y + k * w * h + (count_p-1) * w * h * l;
                    j = x + w * y + k * w * h + count_p * w * h * l;
                    x1 = u1[i] - v1[j];
                    x2 = u2[i] - v2[j];
                    s01 += x1;
                    s02 += x2;
                }

                float norm = sqrtf(s01*s01+s02*s02);

                s1 = norm <= nu ? s01 : (nu * s01 / norm);
                s2 = norm <= nu ? s02 : (nu * s02 / norm);

                for (int k = 0; k < l; k++)
                {
                    i = x + w * y + k * w * h + (count_p-1) * w * h * l;
                    j = x + w * y + k * w * h + count_p * w * h * l;
                    x1 = u1[i] - v1[j];
                    x2 = u2[i] - v2[j];
                    if (k >= k1 && k <= k2) {
                        u1[j] = x1 + (s1 - s01) / K;
                        u2[j] = x2 + (s2 - s02) / K;
                    } else {
                        u1[j] = x1;
                        u2[j] = x2;
                    }
                    u3[j] = u3[i] - v3[j];
                }
                for (int z = 0; z < l; z++)
                {
                    int i = x + w * y + w * h * z + count_p * w * h * l;
                    int j = x + w * y + w * h * z + (count_p-1) * w * h * l;
                    v1[i] = u1[i] - (u1[j] - v1[i]);
                    v2[i] = u2[i] - (u2[j] - v2[i]);
                    v3[i] = u3[i] - (u3[j] - v3[i]);
                }
                count_p++;
            }
        }
    }
}

__global__ void init(float* u,float* ubar,float* p1,float* p2,float* p3,float* f,int w,int h,int l,int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        int I;
        float img;

        for (int c = 0; c < nc; c++) {   
            img = f[x+y*w+c*w*h];

            for (int k = 0; k < l; k++) {
                I = x+y*w+k*w*h+c*w*h*l;
                
                u[I] = img;
                ubar[I] = img;
                p1[I] = 0.f;
                p2[I] = 0.f;
                p3[I] = 0.f;
            }
        }
    }
}

__global__ void isosurface(float* f,float* u,int w,int h,int l,int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h)
    {
        float uk0, uk1, val;
        for (int c = 0; c < nc; c++)
        {
            for (int k = 0; k < l-1; k++)
            {
                uk0 = u[x+y*w+k*h*w+c*h*w*l];
                uk1 = u[x+y*w+(k+1)*h*w+c*h*w*l];
                if (uk0 > 0.5 && uk1 <= 0.5) {
                    val = interpolate(k+1, uk0, uk1, l);
                    break;
                } else {
                    val = uk1;
                }
            }
            f[x+y*w+c*h*w] = val;
        }
    }
}

__global__ void set_y(float* y1, float* y2, float* y3, float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (p-1) * w * h * l;
        y1[i] = u1[j];
        y2[i] = u2[j];
        y3[i] = u3[j];
    }
}

__global__ void set_u_v(float* u1,float* u2,float* u3,float* v1,float* v2,float* v3,float* ubar,float* p1, float* p2, float* p3,float sigma,int w,int h,int l,int p,int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l) {
        float x1, x2, x3, val;

        for (int c = 0; c < nc; c++) {
            int I = x+y*w+z*w*h+c*w*h*l;
            int J;

            val = ubar[I];
            x1 = (x+1<w) ? (ubar[(x+1)+y*w+z*w*h+c*w*h*l] - val) : 0.f;
            x2 = (y+1<h) ? (ubar[x+(y+1)*w+z*w*h+c*w*h*l] - val) : 0.f;
            x3 = (z+1<l) ? (ubar[x+y*w+(z+1)*w*h+c*w*h*l] - val) : 0.f;

            x1 = p1[I] + sigma * x1;
            x2 = p2[I] + sigma * x2;
            x3 = p3[I] + sigma * x3;

            for (int k = 0; k < p; k++) {
                J = x+y*w+z*w*h+k*w*h*l+c*w*h*l*p;

                u1[J] = k < p-1 ? 0.f : x1;
                u2[J] = k < p-1 ? 0.f : x2;
                u3[J] = k < p-1 ? 0.f : x3;

                v1[J] = 0.f;
                v2[J] = 0.f;
                v3[J] = 0.f;
                
            }
        }
    }
}

__global__ void clipping(float* u,float* un,float* p1,float* p2,float* p3,float tau,int w,int h,int l,int nc)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        int I;
        float cur;
        float d1,d2,d3,D;
        for (int c = 0; c < nc; c++)
        {
            I = x+y*w+z*h*w+c*h*w*l;
            cur = u[I];
            un[I] = cur;
            d1 = (y+1<h ? p1[I] : 0.f) - (y>0 ? p1[x+(y-1)*w+z*h*w+c*h*w*l] : 0.f);
            d2 = (x+1<w ? p2[I] : 0.f) - (x>0 ? p2[(x-1)+y*w+z*h*w+c*h*w*l] : 0.f);
            d3 = (z+1<l ? p3[I] : 0.f) - (z>0 ? p3[x+y*w+(z-1)*h*w+c*h*w*l] : 0.f);
            D = cur+tau*(d1+d2+d3);
            if (z==0) {
                u[I]=1.f;
            } else if (z==l-1) {
                u[I]=0.f;
            } else {
                u[I]=fmin(1.f, fmax(0.f, D));
            }
        }
    }
}

__global__ void extrapolate(float* xbar, float* xcur, float* xn, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    int i = x + w * y + w * h * z;

    if (x < w && y < h && z < l) {
        xbar[i] = 2 * xn[i] - xcur[i];
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:

    if (argc <= 2) { cout << "Usage: " << argv[0] << " -i <image> -o <output_image> [-repeats <repeats>] [-gray]" << endl; return 1; }
    
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> -o <output_image> -data <data.txt> -parm <parameter.txt> [-repeats <repeats>] [-gray]" << endl; return 1; }

    // output image
    string output = "";
    bool retO = getParam("o", output, argc, argv);
    if (!retO) cerr << "ERROR: no output image specified" << endl;

    // parameter values
    string parm = "";
    bool ret2 = getParam("parm", parm, argc, argv);
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1000;
    getParam("repeats", repeats, argc, argv);

    // number of computation repetitions to get a better run time measurement
    int dykstra = 10;
    getParam("dykstra", dykstra, argc, argv);
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    int level = 16;
    getParam("level", level, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float L = sqrtf(12);
    getParam("L", L, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    // float tau = 1000;
    float tau = 1.f/L;
    getParam("tau", tau, argc, argv);
    
    // load the input image as grayscale if "-gray" is specifed
    float sigma = 1.f/(L*L*tau);
    getParam("sigma", sigma, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float lambda = 0.1;
    getParam("lambda", lambda, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float nu = 0.01f;
    getParam("nu", nu, argc, argv);
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int dim = w*h*nc;
    int size = w*h*nc*level;
    int projections = level*(level-1)/2 + level + 2;
    // int projections = level * (level+1) / 2 + 1 + 1;
    int nbytes = size*sizeof(float);
    int nbyted = dim*sizeof(float);
    int nbytep = projections*size*sizeof(float);

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    float nrj = 0.f;

    // allocate raw input image array
    // allocate raw input image array
    float* h_u = new float[(size_t)size];
    float* h_un = new float[(size_t)size];
    float* h_img  = new float[(size_t)dim];

    // allocate raw input image for GPU
    float* d_f; hipMalloc(&d_f, nbyted); CUDA_CHECK;
    // float* d_imgOut;hipMalloc(&d_imgOut, nbyted); CUDA_CHECK;

    float* d_u; hipMalloc(&d_u, nbytes); CUDA_CHECK;
    float* d_ubar; hipMalloc(&d_ubar, nbytes); CUDA_CHECK;
    float* d_un; hipMalloc(&d_un, nbytes); CUDA_CHECK;

    float* d_p1; hipMalloc(&d_p1, nbytes); CUDA_CHECK;
    float* d_p2; hipMalloc(&d_p2, nbytes); CUDA_CHECK;
    float* d_p3; hipMalloc(&d_p3, nbytes); CUDA_CHECK;

    float* d_u1; hipMalloc(&d_u1, nbytep); CUDA_CHECK;
    float* d_u2; hipMalloc(&d_u2, nbytep); CUDA_CHECK;
    float* d_u3; hipMalloc(&d_u3, nbytep); CUDA_CHECK;

    float* d_v1; hipMalloc(&d_v1, nbytep); CUDA_CHECK;
    float* d_v2; hipMalloc(&d_v2, nbytep); CUDA_CHECK;
    float* d_v3; hipMalloc(&d_v3, nbytep); CUDA_CHECK;

    size_t available, total;
    hipMemGetInfo(&available, &total);

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (h_img, mIn);

    // copy host memory
    hipMemcpy(d_f, h_img, nbyted, hipMemcpyHostToDevice); CUDA_CHECK;

    // launch kernel
    dim3 block = dim3(32, 8, 4);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (level + block.z - 1) / block.z);
    dim3 block_iso = dim3(32, 8, 1);
    dim3 grid_iso = dim3((w + block_iso.x - 1) / block_iso.x, (h + block_iso.y - 1) / block_iso.y, 1);

    Timer timer; timer.start();

    int count_p = projections;
    int iter;

    init <<<grid_iso, block_iso>>> (d_u,d_ubar,d_p1,d_p2,d_p3,d_f,w,h,level,nc);
    for (iter = 0; iter < repeats; iter++)
    {
        set_u_v <<<grid, block>>> (d_u1,d_u2,d_u3,d_v1,d_v2,d_v3,d_ubar,d_p1,d_p2,d_p3,sigma,w,h,level,projections,nc);
        for (int j = 0; j < dykstra; j++)
        {            
            count_p = 1;
            // set_u_zero <<<grid, block>>> (d_u1, d_u2, d_u3, w, h, level, projections);
            project_on_parabola <<<grid, block>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, d_f, L, lambda, count_p, w, h, level);
            // update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
            // count_p++;
            
            // for (int k1 = 0; k1 < level; k1++)
            // {
            //     for (int k2 = k1; k2 < level; k2++)
            //     {
                    soft_shrinkage <<<grid_iso, block_iso>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, nu, 0, 0, count_p, w, h, level);
                    // update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
                    // count_p++;
            //     }
            // }
        }
        set_y <<<grid, block>>> (d_p1, d_p2, d_p3, d_u1, d_u2, d_u3, w, h, level, projections);
        clipping <<<grid, block>>> (d_u,d_un,d_p1,d_p2,d_p3,tau,w,h,level,nc);
        // hipMemcpy(h_u, d_u, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // hipMemcpy(h_un, d_un, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        // nrj = 0.f;
        // for (int i = 0; i < size; i++)
        // {
        //     nrj += fabs(h_u[i] - h_un[i]);
        // }
        // if (nrj/(w*h*level) <= 5*1E-5) break;
        extrapolate <<<grid, block>>> (d_ubar, d_un, d_u, w, h, level);
    }
    isosurface <<<grid_iso, block_iso>>> (d_f,d_u,w,h,level,nc);
    
    timer.end();  float t = timer.get();  // elapsed time in seconds

    hipMemcpy(h_img, d_f, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;
    // dualEnergy(data, h_energy, count);
    if (!ret2) {
        parameterToConsole(parm,repeats,gray,level,tau,sigma,lambda,nu,w,h,nc,available,total,t,iter);
    } else {
        parameterToFile(parm,repeats,gray,level,tau,sigma,lambda,nu,w,h,nc,available,total,t,iter);
    }

    // free GPU memory
    hipFree(d_f); CUDA_CHECK;
    // hipFree(d_f); CUDA_CHECK;
    
    hipFree(d_u); CUDA_CHECK;
    hipFree(d_ubar); CUDA_CHECK;
    hipFree(d_un); CUDA_CHECK;

    hipFree(d_p1); CUDA_CHECK;
    hipFree(d_p2); CUDA_CHECK;
    hipFree(d_p3); CUDA_CHECK;

    hipFree(d_u1); CUDA_CHECK;
    hipFree(d_u2); CUDA_CHECK;
    hipFree(d_u3); CUDA_CHECK;

    hipFree(d_v1); CUDA_CHECK;
    hipFree(d_v2); CUDA_CHECK;
    hipFree(d_v3); CUDA_CHECK;

    // show input image
    // showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, h_img);
    // showImage("Output", mOut, 100+w+40, 100);

    // wait for key inputs
    // cv::waitKey(0);

    // save input and result
    // cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite(output, mOut*255.f);

    // free allocated arrays
    delete[] h_u;
    delete[] h_un;
    delete[] h_img;
    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
